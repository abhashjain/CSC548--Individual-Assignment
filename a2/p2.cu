#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>

#define ITER_PER_THREAD 256
#define NUMBER_OF_THREAD 256

__global__ void pi_cal(long *niter, long *a,hiprandState *state){
	long idx = (blockDim.x * blockIdx.x) + threadIdx.x; 
	long count = 0;
	float x,y,z;
	hiprand_init(110919, idx , 0, &state[idx]);
	for(long i=0;i<ITER_PER_THREAD;i++){
		x = hiprand_uniform(&state[idx]);
		y = hiprand_uniform(&state[idx]);
		z = (x*x)+(y*y);
		if((idx*ITER_PER_THREAD+i) < (*niter) && z<=1.0f){
			count+=1;
		}
	}
	a[idx] = count;
}


int main(int argc,char **argv){
	if(argc!=2){
		printf("Usage: <exefile> <number_of_iteration>\n");
		return 0;
	}
	long niter = atol(argv[1]);
	int number_of_blocks = ceil(((double)(niter)/(ITER_PER_THREAD*NUMBER_OF_THREAD)));
	double pi =0.0;
	//printf("number of_block %d\n",number_of_blocks);
	long m_size  = number_of_blocks * NUMBER_OF_THREAD;
	long *final_memory = (long *) malloc(m_size*sizeof(long));
	memset(final_memory,0,m_size*sizeof(long));

	long *d_a;
	long *d_iter;
	hiprandState *d_states;

	//allocate memory for current state to generate random number on device
	hipMalloc((void **)&d_a,m_size*sizeof(long));
	hipMalloc((void **)&d_states,m_size*sizeof(hiprandState));
	hipMalloc((void **)&d_iter, sizeof(long));
	
	//copy the value of niter to device iter
	hipMemcpy(d_iter,&niter,sizeof(long),hipMemcpyHostToDevice);

	//invoke gpu kernal program for pi calculation
	pi_cal<<<number_of_blocks,NUMBER_OF_THREAD>>>(d_iter,d_a,d_states);

	//copy the result from d_a to hosts final_memory array
	hipMemcpy(final_memory,d_a,m_size*sizeof(long),hipMemcpyDeviceToHost);
	
	//Now sum all the values in host final_memory to pi	
	for(long i=0;i<m_size;i++){
		pi+=final_memory[i];
	}
	//printf("m_size %d pi %lf\n",m_size,pi);
	//divide the final pi value by size
	pi = (double)(4.0 * pi) / (double)niter;
	//pi /= m_size;

	printf("# of trials= %d , estimate of pi is %.16lf \n",niter,pi);

	//free the allocated memory
	free(final_memory);
	hipFree(d_a);
	hipFree(d_states);
	hipFree(d_iter);
	return 0;
}
